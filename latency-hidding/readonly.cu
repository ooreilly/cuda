#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "readonly.cuh"
#include "helper.cuh"

int main(int argc, char **argv) {

        if (argc != 3) { 
                fprintf(stderr, "usage: %s <number of elements> <shared memory size in bytes> \n", argv[0]);
                exit(-1);
        }

        size_t n = (size_t)atof(argv[1]);
        size_t shared_mem_bytes = (size_t)atof(argv[2]);


        {
                float *u;
                size_t num_bytes = sizeof(u) * n;
                hipMalloc((void**)&u, num_bytes);
                hipMemset(u, 0, num_bytes);

                dim3 threads ( 64, 1, 1);
                dim3 blocks ( (n - 1) / threads.x + 1, 1, 1);

                int maxbytes = 65536;  // 64 KB
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_baseline<float>),
                    hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
                int carveout = cudaSharedmemCarveoutMaxShared;
                cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_baseline<float>),
                    hipFuncAttributePreferredSharedMemoryCarveout, carveout));

                readonly_baseline<float><<<blocks, threads, shared_mem_bytes>>>(u, n);
                hipFree(u);
        }
        
        {
                float *u;
                size_t num_bytes = sizeof(u) * n;
                hipMalloc((void**)&u, num_bytes);
                hipMemset(u, 0, num_bytes);

                const int unroll = 4;
                dim3 threads ( 64, 1, 1);
                dim3 blocks ( (n / unroll - 1) / threads.x + 1, 1, 1);

                int maxbytes = 65536;  // 64 KB
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_unroll<unroll>),
                    hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
                int carveout = cudaSharedmemCarveoutMaxShared;
                cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_unroll<unroll>),
                    hipFuncAttributePreferredSharedMemoryCarveout, carveout));

                readonly_unroll<unroll><<<blocks, threads, shared_mem_bytes>>>(u, n);
                hipFree(u);
        }

        // Float 4 kernel
        {
                float4 *u;
                assert(n % 4 == 0);
                size_t n4 = n / 4;
                size_t num_bytes = sizeof(float4) * n4;
                cudaErrCheck(hipMalloc((void**)&u, num_bytes));
                cudaErrCheck(hipMemset(u, 0, num_bytes));

                dim3 threads ( 64, 1, 1);
                dim3 blocks ( (n4 - 1) / threads.x + 1, 1, 1);

                int maxbytes = 65536;  // 64 KB
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_float4),
                    hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
                int carveout = cudaSharedmemCarveoutMaxShared;
                cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_float4),
                    hipFuncAttributePreferredSharedMemoryCarveout, carveout));

                readonly_float4<<<blocks, threads, shared_mem_bytes>>>(u, n4);
                hipFree(u);

        }

}
