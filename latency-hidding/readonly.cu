#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "readonly.cuh"
#include "helper.cuh"

const int CLOCK_LATENCY = 80;

template <typename T>
T max(T *a, int n) {
        int best = 0;
        for (int i = 0; i < n; ++i)
                best = best < a[i] ? a[i] : best;
        return best;
}

template <typename T>
T average(T *a, int n) {
        int avg = 0;
        for (int i = 0; i < n; ++i)
                avg += a[i];
        return avg / n;
}

int main(int argc, char **argv) {

        if (argc != 3) { 
                fprintf(stderr, "usage: %s <number of elements> <shared memory size in bytes> \n", argv[0]);
                exit(-1);
        }

        size_t n = (size_t)atof(argv[1]);
        size_t shared_mem_bytes = (size_t)atof(argv[2]);


        {
                float *u;
                size_t num_bytes = sizeof(u) * n;
                unsigned int *h_duration, *d_duration;
                hipMalloc((void**)&u, num_bytes);
                hipMalloc((void**)&d_duration, sizeof(unsigned int) * n);
                h_duration = (unsigned int*)malloc(sizeof(unsigned int) * n);
                hipMemset(u, 0, num_bytes);

                dim3 threads ( 64, 1, 1);
                dim3 blocks ( (n - 1) / threads.x + 1, 1, 1);

                int maxbytes = 65536;  // 64 KB
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_baseline<float>),
                    hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
                int carveout = cudaSharedmemCarveoutMaxShared;
                cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_baseline<float>),
                    hipFuncAttributePreferredSharedMemoryCarveout, carveout));

                readonly_baseline<float><<<blocks, threads, shared_mem_bytes>>>(u, n, d_duration);
                hipFree(u);

                hipMemcpy(h_duration, d_duration, sizeof(unsigned int) * n, hipMemcpyDeviceToHost);
                printf("latency: %d \n", max(h_duration, n) - CLOCK_LATENCY);
        }
        
        {
                float *u;
                size_t num_bytes = sizeof(u) * n;
                hipMalloc((void**)&u, num_bytes);
                hipMemset(u, 0, num_bytes);

                const int unroll = 4;
                dim3 threads ( 64, 1, 1);
                dim3 blocks ( (n / unroll - 1) / threads.x + 1, 1, 1);

                int maxbytes = 65536;  // 64 KB
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_unroll<unroll>),
                    hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
                int carveout = cudaSharedmemCarveoutMaxShared;
                cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_unroll<unroll>),
                    hipFuncAttributePreferredSharedMemoryCarveout, carveout));

                readonly_unroll<unroll><<<blocks, threads, shared_mem_bytes>>>(u, n);
                hipFree(u);
        }

        // Float 4 kernel
        {
                float4 *u;
                assert(n % 4 == 0);
                size_t n4 = n / 4;
                size_t num_bytes = sizeof(float4) * n4;
                cudaErrCheck(hipMalloc((void**)&u, num_bytes));
                cudaErrCheck(hipMemset(u, 0, num_bytes));

                dim3 threads ( 64, 1, 1);
                dim3 blocks ( (n4 - 1) / threads.x + 1, 1, 1);

                int maxbytes = 65536;  // 64 KB
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_float4),
                    hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
                int carveout = cudaSharedmemCarveoutMaxShared;
                cudaErrCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    readonly_float4),
                    hipFuncAttributePreferredSharedMemoryCarveout, carveout));

                readonly_float4<<<blocks, threads, shared_mem_bytes>>>(u, n4);
                hipFree(u);

        }

}
